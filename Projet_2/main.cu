#include <iostream>
#include <cstdlib>
#include <iomanip>

#include "utils/chronoCPU.hpp"
#include "utils/chronoGPU.hpp"
#include "los/ppm.hpp"
#include "carteCPU.hpp"
#include "carteGPU.hpp"

#define Cx 245
#define Cy 497

int main(int argc, char **argv)
{
	// Parse program arguments
	// ================================================================================================================
	// Allocation and initialization

	// ================================================================================================================

	// ================================================================================================================
	// CPU sequential
	std::cout << "============================================" << std::endl;
	std::cout << "         Sequential version on CPU          " << std::endl;
	std::cout << "============================================" << std::endl;



    Heightmap h_inCPU("img/input/1.input.ppm");
    Heightmap h_outCPU(h_inCPU.getWidth(), h_inCPU.getHeight());

	ChronoGPU chrCPU;
	chrCPU.start();		// CPU method
	drawMap(h_inCPU.getPtr(), h_outCPU.getPtr(), h_inCPU.getWidth(), h_inCPU.getHeight(),  Cx, Cy);
	//void drawMap(int *data,uint8_t *h_data, Heightmap h_out, const int MapSize, const int MapWidth, const int MapHeight,const int Cx,const int Cy)
	h_outCPU.saveTo("img/Result/CPU/LimousinCPU.ppm");
	chrCPU.stop();

	const float timeComputeCPU = chrCPU.elapsedTime();
	std::cout << "-> Done : " << std::fixed << std::setprecision(2) << timeComputeCPU << " ms" << std::endl
			  << std::endl;

	// ================================================================================================================

	// ================================================================================================================
	// GPU CUDA
	std::cout << "============================================" << std::endl;
	std::cout << "          Parallel version on GPU           " << std::endl;
	std::cout << "============================================" << std::endl;

	// data GPU

    Heightmap h_inGPU("img/input/1.input.ppm");
    Heightmap h_outGPU(h_inGPU.getWidth(), h_inGPU.getHeight());

	// data GPU

	// GPU allocation
	
	carteGPU(h_inGPU.getPtr(), h_outGPU.getPtr(), h_inGPU.getWidth(), h_inGPU.getHeight(), Cx, Cy);
	h_outGPU.saveTo("img/Result/GPU/LimousinGPU.ppm");
	// ======================

	/*const float timeAllocGPU = chrCPU.elapsedTime();
	std::cout << "-> Done : " << std::fixed << std::setprecision(2) << timeAllocGPU << " ms" << std::endl
			  << std::endl;*/

	// Copy from host to device
	

	
	// Launch kernel

	// copy from device to host

	// Free GPU memory

	// ================================================================================================================

	std::cout << "============================================" << std::endl;
	std::cout << "              Checking results              " << std::endl;
	std::cout << "============================================" << std::endl;

	for (int i = 0; i < h_inCPU.getHeight(); i++)
	{
		for (int j = 0; i < h_inCPU.getWidth(); j++)
			{
				if (h_outCPU.getPixel(j, i) != h_outGPU.getPixel(j, i))
				{
					std::cout << "error on index (" << i << ", " << j << ")" << std::endl;
					std::cout << "value CPU : " << +h_outCPU.getPixel(j, i) << ", value GPU : " << +h_outGPU.getPixel(j, i) << std::endl;

					return EXIT_FAILURE;
				} 
			}
	}



	/*std::cout << "Congratulations! Job's done!" << std::endl
			  << std::endl;

	std::cout << "============================================" << std::endl;
	std::cout << "            Times recapitulation            " << std::endl;
	std::cout << "============================================" << std::endl;
	std::cout << "-> CPU	Sequential" << std::endl;
	std::cout << "   - Computation:    " << std::fixed << std::setprecision(2)
			  << timeComputeCPU << " ms" << std::endl;
	std::cout << "-> GPU	" << std::endl;
	std::cout << "   - Allocation:     " << std::fixed << std::setprecision(2)
			  << timeAllocGPU << " ms " << std::endl;
	std::cout << "   - Host to Device: " << std::fixed << std::setprecision(2)
			  << timeHtoDGPU << " ms" << std::endl;
	std::cout << "   - Computation:    " << std::fixed << std::setprecision(2)
			  << timeComputeGPU << " ms" << std::endl;
	std::cout << "   - Device to Host: " << std::fixed << std::setprecision(2)
			  << timeDtoHGPU << " ms " << std::endl
			  << std::endl;
*/
	return EXIT_SUCCESS;
}