#include "hip/hip_runtime.h"
#include "utils/commonCUDA.hpp"
#include "utils/chronoGPU.hpp"

#include "carteGPU.hpp"
#include <cmath>
#include <iostream>
#include <cstdlib>
#include <iomanip>

using namespace std;

__global__ void kernelMap(uint8_t *h_in, uint8_t *h_out, const int MapWidth, const int MapHeight, const int Cx, const int Cy)
{
    
    float Dx, Dy, Dz,  D;
    float angle, angle_ref;
    float Cx_dda, Cy_dda; 
    float incX, incY;
    int Lx, Ly;
    for(int indexY = blockDim.y * blockIdx.y + threadIdx.y; indexY < MapHeight; indexY += blockDim.y * gridDim.y)
    {
        for(int indexX = blockDim.x * blockIdx.x + threadIdx.x; indexX < MapWidth; indexX += blockDim.x * gridDim.x)
        {
        // DDA entre le point c (Cx, Cy) et le point P (indexX, indexY);
            
            Dx = indexX - Cx;   // delta x
            Dy = indexY - Cy;   // delta y
            Dz = h_in[indexY * MapWidth + indexX] - h_in[Cy * MapWidth + Cx];   // delta z
            D = max(abs(Dx), abs(Dy));  // delta positif max entre Dx et Dy

            angle_ref = atan(Dz / sqrt((Dx * Dx) + (Dy * Dy)));
            Cx_dda = (float)Cx , Cy_dda = (float)(Cy);
            incX = Dx / D;
            incY = Dy / D;
            

            h_out[indexY * MapWidth + indexX] = 244;
            for (int i = 0; i < D - 1; i++)
            {
                Cx_dda += incX;
                Cy_dda += incY;
                Lx = (int)(round(Cx_dda));
                Ly = (int)(round(Cy_dda));

                Dx = indexX - Lx;
                Dy = indexY - Ly;    
                Dz = h_in[indexY * MapWidth + indexX] - h_in[Ly * MapWidth + Lx];  


                // Calcule Angle 

                angle = atan(Dz / sqrt((Dx * Dx) + (Dy * Dy)));     

                if (angle_ref >= angle)
                {
                    h_out[indexY * MapWidth + indexX] = 0;
                    break;
                }                         
            } 
        }
    }
}



void carteGPU(uint8_t *h_in, uint8_t *h_out, const int MapWidth, const int MapHeight, const int Cx, const int Cy)
{
    ChronoGPU chrGPU;
    uint8_t *dev_h_in;
    uint8_t *dev_h_out;

    size_t size = MapWidth * MapHeight * sizeof(uint8_t);

    hipMalloc((void**) &dev_h_in, size);
    hipMalloc((void**) &dev_h_out, size);

    hipMemcpy(dev_h_in, h_in, size, hipMemcpyHostToDevice);


    dim3 gridDim(16, 16);
    dim3 blockDim(16, 16);


    kernelMap<<<gridDim, blockDim>>>(dev_h_in, dev_h_out, MapHeight, MapHeight, Cx, Cy);

    hipMemcpy(h_out, dev_h_out, size, hipMemcpyDeviceToHost);

    hipFree(dev_h_in);
    hipFree(dev_h_out);


}